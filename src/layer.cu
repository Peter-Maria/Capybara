#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include "externel_headers.cuh"
#include "layer.cuh"

template<class computeType>
void layer<computeType>::allocIOMemory()
{
  int inSize  = batch_size * channels_in * width_in * height_in * sizeof( computeType );
  int outSize = batch_size * channels_out * width_out * height_out * sizeof( computeType );

  if( first_pr ) {
	HandleCudaError( hipMalloc( &srcPtr,     inSize ) );
	HandleCudaError( hipMalloc( &srcDiffPtr, inSize ) );
  }
  HandleCudaError( hipMalloc( &dstPtr, outSize ) );
  HandleCudaError( hipMalloc( &dstDiffPtr, outSize ) );

  if( last_pr ) {
    switch( learning_problem ) {
      case classification:
    	HandleCudaError( hipMalloc( &tGroundTruth, batch_size * sizeof( computeType ) ) );
        break;
      case regression:
    	HandleCudaError( hipMalloc( &tGroundTruth, outSize ) );
        break;
    }
  }
}

template<class computeType>
void layer<computeType>::freeIOMemory()
{
  if( first_pr ) {
	if( srcPtr ) {
	  HandleCudaError( hipFree( srcPtr ) );
	}
	if( srcDiffPtr ) {
      HandleCudaError( hipFree( srcDiffPtr ) );
	}
  }
  if( dstPtr ) {
	HandleCudaError( hipFree( dstPtr ) );
  }
  if( dstDiffPtr ) {
	HandleCudaError( hipFree( dstDiffPtr ) );
  }
  if( last_pr ) {
	if( tGroundTruth ) {
      HandleCudaError( hipFree( tGroundTruth ) );
	}
  }
}

template<class computeType>
bool layer<computeType>::adaptable_weight()
{
  return adaptable_weight_pr;
}

template<class computeType>
__global__ void computeDifferenceClassificationKernel( computeType* prediction, computeType* groundTruth,
		                                               int batch_size, int nr_classes, double scale )
{
  int gidx = blockDim.x * blockIdx.x + threadIdx.x;
  if( gidx < batch_size ) {
	int position = (int) groundTruth[ gidx ];
	prediction[ gidx*nr_classes + position ] -= (computeType) 1.0;
	for( int i = 0; i<nr_classes; i++ ) {
	  prediction[ gidx*nr_classes + i ] *= scale;
	}
  }
}

template<class computeType>
__global__ void computeDifferenceRegressionKernel( computeType* prediction, computeType* groundTruth,
		                                           int batch_size, int nr_classes, double scale )
{
  int gidx = blockDim.x * blockIdx.x + threadIdx.x;
  if( gidx < batch_size * nr_classes ) {
    prediction[ gidx ] = scale * ( groundTruth[ gidx ] - prediction[ gidx ] );
  }
}

template<class computeType>
void layer<computeType>::compute_difference()
{
  int nBlocks    = 0;
  int nr_classes = channels_out * width_out * height_out;

  HANDLE_CUDA_ERROR( hipMemcpy( dstDiffPtr, dstPtr, nr_classes * batch_size * sizeof( computeType ) ) );

  if( learning_problem == classification ) {
    nBlocks = ( batch_size + tperblock - 1 ) / tperblock;
    computeDifferenceClassificationKernel<computeType><<<nBlocks,tperblock>>>( dstDiffPtr, tGroundTruth,
    		                                                                   batch_size, nr_classes, scale_diff );
  } else {
	nBlocks = ( batch_size * nr_classes + tperblock - 1 ) / tperblock;
	computeDifferenceRegressionKernel<computeType><<<nBlocks,tperblock>>>( dstDiffPtr, tGroundTruth,
                                                                           batch_size, nr_classes, scale_diff );
  }
}

template<class computeType>
bool layer<computeType>::first()
{
  return first_pr;
}

template<class computeType>
bool layer<computeType>::last()
{
  return last_pr;
}

template<class computeType>
layer<computeType>::layer()
{
  srcPtr       = NULL;
  dstPtr       = NULL;
  srcDiffPtr   = NULL;
  dstDiffPtr   = NULL;
  tGroundTruth = NULL;

  channels_in  = -1;
  width_in     = -1;
  height_in    = -1;

  channels_out = -1;
  width_out    = -1;
  height_out   = -1;

  tperblock           = 512;
  batch_size          = -1;
  adaptable_weight_pr = false;

  learning_problem    = classification;
  last_pr             = false;
  first_pr            = false;

  scale_diff          = 5e-4;
}
